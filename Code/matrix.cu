#include <stdio.h>

#include "matrix.h"
#include "kernel_call.h"

void matrixMultiplication(float *A, float *B, float *C, int N);
void matrixAddition(float *A, float *B, float *C, int N);
void matrix_float_Addition(float *A, float num, float *C, int N);

Matrix::Matrix(int N)
{
    hipError_t result = hipMalloc(&matrix_d, N * N * sizeof(float));
    if (result != hipSuccess)
    {
        printf("Error: failed to allocate memory on device (%s).\n", hipGetErrorString(result));
        exit(1);
    }
    size = N;
}

Matrix::Matrix(const Matrix& other)
{
    size = other.size;
    hipError_t result = hipMalloc(&matrix_d, size * size * sizeof(float));
    if (result != hipSuccess)
    {
        printf("Error: failed to allocate memory on device (%s).\n", hipGetErrorString(result));
        exit(1);
    }
    result = hipMemcpy(matrix_d, other.matrix_d, size * size * sizeof(float), hipMemcpyDeviceToDevice);
    if (result != hipSuccess)
    {
        printf("Error: failed to copy device memory (%s).\n", hipGetErrorString(result));
        exit(1);
    }
}

Matrix::~Matrix()
{
    hipError_t result = hipFree(matrix_d);
    if (result != hipSuccess)
    {
        printf("Error: failed to free device memory (%s).\n", hipGetErrorString(result));
    }
}

Matrix& Matrix::operator=(const Matrix& other)
{
    if (this != &other)
    {
        if (matrix_d)
        {
            hipError_t result = hipFree(matrix_d);
            if (result != hipSuccess)
            {
                printf("Error: failed to free device memory (%s).\n", hipGetErrorString(result));
            }
        }

        size = other.size;
        hipError_t result = hipMalloc(&matrix_d, size * size * sizeof(float));
        if (result != hipSuccess)
        {
            printf("Error: failed to allocate memory on device (%s).\n", hipGetErrorString(result));
            exit(1);
        }
        result = hipMemcpy(matrix_d, other.matrix_d, size * size * sizeof(float), hipMemcpyDeviceToDevice);
        if (result != hipSuccess)
        {
            printf("Error: failed to copy device memory (%s).\n", hipGetErrorString(result));
            exit(1);
        }
    }
    return *this;
}

Matrix Matrix::operator*(Matrix const& matrix)
{
    if (matrix.size != size)
    {
        printf("Error: can't multiply two matrices of different sizes\n");
        exit(1);
    }
    
    Matrix res(size);
    matrixMultiplication(matrix_d, matrix.matrix_d, res.matrix_d, size);

    return res;
}

Matrix Matrix::operator+(Matrix const& matrix)
{
    if (matrix.size != size)
    {
        printf("Error: can't add two matrices of different sizes\n");
        exit(1);
    }
    
    Matrix res(size);
    matrixAddition(matrix_d, matrix.matrix_d, res.matrix_d, size);

    return res;
}

Matrix Matrix::operator+(float num)
{
    Matrix res(size);
    matrix_float_Addition(matrix_d, num, res.matrix_d, size);

    return res;
}

float Matrix::get(int x, int y)
{
    float element;
    hipError_t result = hipMemcpy(&element, matrix_d + size * y + x, sizeof(float), hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        printf("Error: failed to copy to host memory (%s)\n", hipGetErrorString(result));
        exit(1);
    }
    
    return element;
}

void Matrix::set(int x, int y, float value)
{
    hipError_t result = hipMemcpy(matrix_d + size * y + x, &value, sizeof(float), hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Error: failed to copy to device memory (%s)\n", hipGetErrorString(result));
        exit(1);
    }
}