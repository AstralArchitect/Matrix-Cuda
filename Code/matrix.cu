#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "matrix.h"
#include "kernel_call.h"

Matrix::Matrix(unsigned int dimX, unsigned int dimY)
{
    hipError_t result = hipMalloc((void**)&matrix_d, dimX * dimY * sizeof(double));
    if (result != hipSuccess)
    {
        printf("Error: failed to allocate memory on device (%s).\n", hipGetErrorString(result));
        exit(1);
    }
    size.x = dimX;
    size.y = dimY;
}

Matrix::Matrix(const Matrix& other)
{
    size.x = other.size.x;
    size.y = other.size.y;
    hipError_t result = hipMalloc((void**)&matrix_d, size.x * size.y * sizeof(double));
    if (result != hipSuccess)
    {
        printf("Error: failed to allocate memory on device (%s).\n", hipGetErrorString(result));
        exit(1);
    }
    result = hipMemcpy(matrix_d, other.matrix_d, size.x * size.y * sizeof(double), hipMemcpyDeviceToDevice);
    if (result != hipSuccess)
    {
        printf("Error: failed to copy device memory (%s).\n", hipGetErrorString(result));
        exit(1);
    }
}

Matrix::~Matrix()
{
    hipError_t result = hipFree(matrix_d);
    if (result != hipSuccess)
    {
        printf("Error: failed to free device memory (%s).\n", hipGetErrorString(result));
    }
}

Matrix& Matrix::operator=(const Matrix& other)
{
    if (this != &other)
    {
        if (matrix_d)
        {
            hipError_t result = hipFree(matrix_d);
            if (result != hipSuccess)
            {
                printf("Error: failed to free device memory (%s).\n", hipGetErrorString(result));
            }
        }

        size = other.size;
        hipError_t result = hipMalloc((void**)&matrix_d, size.x * size.y * sizeof(double));
        if (result != hipSuccess)
        {
            printf("Error: failed to allocate memory on device (%s).\n", hipGetErrorString(result));
            exit(1);
        }
        result = hipMemcpy(matrix_d, other.matrix_d, size.x * size.y * sizeof(double), hipMemcpyDeviceToDevice);
        if (result != hipSuccess)
        {
            printf("Error: failed to copy device memory (%s).\n", hipGetErrorString(result));
            exit(1);
        }
    }
    return *this;
}

Matrix Matrix::operator*(Matrix const& matrix)
{
    if (matrix.size.x != size.x && matrix.size.y != size.y)
    {
        printf("Error: can't multiply two matrices of different sizes\n");
        exit(1);
    }
    
    Matrix res(size.x, size.y);
    matrixMultiplication(matrix_d, matrix.matrix_d, res.matrix_d, size.x, size.y);

    return res;
}

Matrix Matrix::operator+(Matrix const& matrix)
{
    if (matrix.size.x != size.x && matrix.size.y != size.y)
    {
        printf("Error: can't add two matrices of different sizes\n");
        exit(1);
    }
    
    Matrix res(size.x, size.y);
    matrixAddition(matrix_d, matrix.matrix_d, res.matrix_d, size.x, size.y);

    return res;
}

Matrix Matrix::operator+(double num)
{
    Matrix res(size.x, size.y);
    matrix_double_Addition(matrix_d, num, res.matrix_d, size.x, size.y);

    return res;
}

double Matrix::get(int x, int y)
{
    double element;
    hipError_t result = hipMemcpy(&element, matrix_d + size.x * y + x, sizeof(double), hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        printf("Error: failed to copy to host memory (%s)\n", hipGetErrorString(result));
        exit(1);
    }
    
    return element;
}

void Matrix::set(int x, int y, double value)
{
    hipError_t result = hipMemcpy(matrix_d + size.x * y + x, &value, sizeof(double), hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Error: failed to copy to device memory (%s)\n", hipGetErrorString(result));
        exit(1);
    }
}

double *Matrix::getPtr()
{
    return matrix_d;
}