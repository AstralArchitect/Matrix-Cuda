#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <math.h>

#include "kernel.h"

hipblasHandle_t handle;

void initCuBLAS()
{
    // First, create a cuBLAS handle:
    hipblasStatus_t cublasStat = hipblasCreate(&handle);

    // Set the math mode to allow cuBLAS to use Tensor Cores:
    cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
}

void destroyCuBLAS()
{
    hipblasDestroy(handle);
}

void matrixMultiplication(float *A, float *B, float *C, int dimX, int dimY)
{
    /*dim3 threadsPerBlock(8, 8);
    dim3 blocksPerGrid((dimX + threadsPerBlock.x - 1) / threadsPerBlock.x, (dimY + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    Mat2DMult<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, dimX, dimY);
    hipDeviceSynchronize();
    
    return;*/

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimX, dimY, dimY, &alpha,
                 A, HIP_R_32F, dimX,
                 B, HIP_R_32F, dimX,
                 &beta, C, HIP_R_32F, dimX,
                 HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void matrixAddition(float *A, float *B, float *C, int dimX, int dimY)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((dimX + threadsPerBlock.x - 1) / threadsPerBlock.x, (dimY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    Mat2DAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, dimX, dimY);

    hipDeviceSynchronize();
}

void matrix_float_Addition(float *A, float num, float *C, int dimX, int dimY)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((dimX + threadsPerBlock.x - 1) / threadsPerBlock.x, (dimY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    Mat2D_float_Add<<<blocksPerGrid, threadsPerBlock>>>(A, num, C, dimX, dimY);

    hipDeviceSynchronize();
}