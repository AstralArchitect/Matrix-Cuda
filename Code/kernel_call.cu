#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <math.h>

#include "kernel.h"

hipblasHandle_t handle;

void initCuBLAS()
{
    // First, create a cuBLAS handle:
    hipblasStatus_t cublasStat = hipblasCreate(&handle);

    // Set the math mode to allow cuBLAS to use Tensor Cores:
    cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
}

void destroyCuBLAS()
{
    hipblasDestroy(handle);
}

void matrixMultiplication(double *A, double *B, double *C, int dimX, int dimY)
{
    /*dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((dimX + threadsPerBlock.x - 1) / threadsPerBlock.x, (dimY + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    Mat2DMult<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, dimX, dimY);
    hipDeviceSynchronize();
    
    return;*/

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimX, dimY, dimY, &alpha,
                 A, HIP_R_64F, dimX,
                 B, HIP_R_64F, dimX,
                 &beta, C, HIP_R_64F, dimX,
                 HIP_R_64F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void matrixAddition(double *A, double *B, double *C, int dimX, int dimY)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((dimX + threadsPerBlock.x - 1) / threadsPerBlock.x, (dimY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    Mat2DAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, dimX, dimY);

    hipDeviceSynchronize();
}

void matrix_double_Addition(double *A, double num, double *C, int dimX, int dimY)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((dimX + threadsPerBlock.x - 1) / threadsPerBlock.x, (dimY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    Mat2D_double_Add<<<blocksPerGrid, threadsPerBlock>>>(A, num, C, dimX, dimY);

    hipDeviceSynchronize();
}